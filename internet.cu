#include "hip/hip_runtime.h"
#include "common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <stdlib.h>
#define TILESIZE 2

int numARows;   // number of rows in the matrix A
int numAColumns;  // number of columns in the matrix A
int numBRows;   // number of rows in the matrix B
int numBColumns;  // number of columns in the matrix B
int numCRows;  // number of rows in the matrix C (you have to set this)
int numCColumns; // number of columns in the matrix C (you have to set this)


// Compute C = A * B
//*************************************************************
//Kernel for shared memory/ Tiled execution
__global__ void matrixMultiplyShared(float * A, float * B, float * C,
                                    int numARows, int numAColumns,
                                    int numBRows, int numBColumns,
                                    int numCRows, int numCColumns)
{
    __shared__ float sA[TILESIZE][TILESIZE];   // Tile size to store elements in shared memory
    __shared__ float sB[TILESIZE][TILESIZE];

    int Row = blockDim.y*blockIdx.y + threadIdx.y; //To generate ids of threads.
    int Col = blockDim.x*blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((numAColumns - 1)/ TILESIZE) + 1); k++)
    {
        if ( (Row < numARows) && (threadIdx.x + (k*TILESIZE)) < numAColumns)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row*numAColumns) + threadIdx.x + (k*TILESIZE)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if ( Col < numBColumns && (threadIdx.y + k*TILESIZE) < numBRows)//Copy Data to Tile from Matrix (Global Memory to Shared Memory)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k*TILESIZE)*numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < TILESIZE; ++j)//Multiplying Elements present in tile
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < numCRows && Col < numCColumns)//Saving Final result into Matrix C
    {
        C[Row*numCColumns + Col] = Cvalue;
    }
}
//*************************************************************
void Print_Mat(int Row,int Col,float * Mat)//Function To print the Matrix
{
 for(int i=0;i<Row*Col;i++)
   {
   printf("%f  ",*(Mat+i));

   if((i%Col)==0 )
    {
     printf("\n");
    }
   }
}//Function close
//*************************************************************
//Normal CPU Matrix Multiplication
void matMultiplyOnHost(float * A, float * B, float * C, int numARows,
                        int numAColumns, int numBRows, int numBColumns,
                        int numCRows, int numCColumns)
{
    for (int i=0; i < numARows; i ++)
    {
        for (int j = 0; j < numAColumns; j++)
        {
            C[i*numCColumns + j ] = 0.0;
            for (int k = 0; k < numCColumns; k++)
            {
                C[i*numCColumns + j ] += A[i*numAColumns + k] * B [k*numBColumns + j];
            }
        }
    }
    return;
}
//*************************************************************
int main(int argc, char ** argv) {
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * hostComputedC;
    float * deviceA;
    float * deviceB;
    float * deviceC;

    // Please adjust rows and columns according to you need.

    printf("\nPlease Enter Rows and Columns of A:");
    scanf("%d %d",&numARows,&numAColumns);

    printf("\nPlease Enter Rows and Columns of B:");
    scanf("%d %d",&numBRows,&numBColumns);

    hostA = (float *) malloc(sizeof(float)*numARows*numAColumns);
    hostB = (float *) malloc(sizeof(float)*numBRows*numBColumns);

    for (int i = 0; i < numARows*numAColumns; i++)//Matrix Initialization
    {
        hostA[i]=1.0;
    }
    for (int i = 0; i < numBRows*numBColumns; i++)
    {
        hostB[i]=1.0;
    }

    printf("\nMatrix A Values:\n");
    Print_Mat(numARows,numAColumns,hostA);//Function Call

    printf("\n\nMatrix B Values:\n");
    Print_Mat(numBRows,numBColumns,hostB);//Function Call



    // Setting numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;

    hostC = (float *) malloc(sizeof(float)*numCRows*numCColumns);
    hostComputedC = (float *) malloc(sizeof(float)*numCRows*numCColumns);

    // Allocating GPU memory
    SAFE_CALL(hipMalloc((void **)&deviceA, sizeof(float)*numARows*numAColumns));
    SAFE_CALL(hipMalloc((void **)&deviceB, sizeof(float)*numBRows*numBColumns));
    SAFE_CALL(hipMalloc((void **)&deviceC, sizeof(float)*numCRows*numCColumns));

    // Copy memory to the GPU
    SAFE_CALL(hipMemcpy(deviceA, hostA, sizeof(float)*numARows*numAColumns, hipMemcpyHostToDevice));
    SAFE_CALL(hipMemcpy(deviceB, hostB, sizeof(float)*numBRows*numBColumns, hipMemcpyHostToDevice));

    // Initialize the grid and block dimensions

    dim3 dimGrid((numCColumns/TILESIZE) + 1, (numCRows/TILESIZE) + 1, 1);//Number of Blocks required
    dim3 dimBlock(TILESIZE, TILESIZE, 1);//Number of threads in each block

    //@@ Launch the GPU Kernel here
    matrixMultiplyShared<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    hipError_t err1 = hipPeekAtLastError();//To capture last error in function call

    hipDeviceSynchronize();//To synchronize the device

    // Copy the results in GPU memory back to the CPU
    SAFE_CALL(hipMemcpy(hostC, deviceC, sizeof(float)*numCRows*numCColumns, hipMemcpyDeviceToHost));

    printf("\nMatrix C From Device\n");
    Print_Mat(numCRows,numCColumns,hostC);//Function Call

    matMultiplyOnHost(hostA, hostB, hostComputedC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

    printf("\nMatrix C From Host\n");
    Print_Mat(numCRows,numCColumns,hostComputedC);//Function Call

    for (int i=0; i < numCColumns*numCRows; i++)//Compare both the result matrices 1. MatrixMultiplyonHost 2. MatrixMultiplyonDevice
    {
        if (hostComputedC[i]  != hostC[i] )
        {
            printf("Mismatch at Row = %d Col = %d hostComputed[] = %f --device[] %f\n", i / numCColumns, i % numCColumns, hostComputedC[i], hostC[i]);
            break;
        }
    }

    printf("\n Number of Blocks Created:%d \n",((numCColumns/TILESIZE) + 1)*((numCColumns/TILESIZE) + 1));
    printf("\n Number of Threads Per Block: %d \n",(TILESIZE*TILESIZE));

    // Free the GPU memory
    SAFE_CALL(hipFree(deviceA));
    SAFE_CALL(hipFree(deviceB));
    SAFE_CALL(hipFree(deviceC));
    //Free the Pointer Memory
    free(hostA);
    free(hostB);
    free(hostC);
    free(hostComputedC);

    return 0;
}
